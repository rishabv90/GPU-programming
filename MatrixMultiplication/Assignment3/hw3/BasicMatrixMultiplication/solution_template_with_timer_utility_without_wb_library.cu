#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
  //@@ Insert code to implement basic matrix multiplication for
  //@@ arbitrary size using global memory. 
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)


  int dim_param;

hipEvent_t astartEvent, astopEvent;
float aelapsedTime;
hipEventCreate(&astartEvent);
hipEventCreate(&astopEvent);

  args = wbArg_read(argc, argv);

 //ali  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = 0;  // set to correct value
  numCColumns = 0;  // set to correct value
  //@@ Allocate the hostC matrix
  
 //ali  wbTime_stop(Generic, "Importing data and creating memory on host");

//ali  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
//ali  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
//ali  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
 //ali  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here for A, B and C
  
  
 //ali wbTime_stop(GPU, "Allocating GPU memory.");

 //ali wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here for A and B
  
  
 //ali wbTime_stop(GPU, "Copying input memory to the GPU.");

  
  
  // ali: this loop will sweep blocks 4x4, 8x8, 16x16 and 32x32
  dim_param=4;
  for(dim_param=4; dim_param<33; dim_param=dim_param*2) {
    dim3 DimBlock(dim_param,dim_param,1);
    //@@ Initialize the grid dimensions here
    // use dim3 structure for setting grid dimensions
	dim3 DimGrid();  // need to fill in DimGrid() call
    //ali  wbTime_start(Compute, "Performing CUDA computation");
    hipEventRecord(astartEvent, 0);
   //@@ Launch the GPU Kernel here
  
  hipDeviceSynchronize();
  //hipDeviceSynchronize();

  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("Total execution time (ms) %f for block size %d x %d matrix size of %d x %d and %d x %d\n",aelapsedTime,dim_param,dim_param,numARows,numAColumns,numBRows,numBColumns);

  //ali  wbTime_stop(Compute, "Performing CUDA computation");

  // wbLog(TRACE, "The block dimensions are ", dim_param, " x ", dim_param);
  //ali  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  
  //ali  wbTime_stop(Copy, "Copying output memory to the CPU");
  wbSolution(args, hostC, numCRows, numCColumns);
} /* end of block size sweep */
//ali  wbTime_start(GPU, "Freeing GPU Memory");

//@@ Free the GPU memory here
 
//ali  wbTime_stop(GPU, "Freeing GPU Memory");
//ali  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
