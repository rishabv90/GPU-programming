#include "hip/hip_runtime.h"
#include <wb.h>

//just testing
/*
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
*/

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  //   and launch your kernel from the main function

	int threadId;

	threadId = blockDim.x * blockIdx.x + threadIdx.x;

	if(threadId < len){
		
		out[threadId] = in1[threadId] + in2[threadId]; 
		printf("\n SUM for tid %d , bid %d is %f + %f = %f\n", threadIdx.x, blockIdx.x, in1[threadId],  in2[threadId], out[threadId]);
		
	}
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here - device input1 + 2 and output
  if(hipMalloc((void **) &deviceInput1, inputLength*sizeof(float)) != hipSuccess){
	printf("Malloc error for device_input1");
	return 0;
  }  

  if(hipMalloc((void **) &deviceInput2, inputLength*sizeof(float)) != hipSuccess){
	printf("Malloc error for device_input2");
	return 0;
  }  
 	 

  if(hipMalloc((void **) &deviceOutput, inputLength*sizeof(float)) != hipSuccess){
	printf("Malloc error for deviceOutput");
	return 0;
  }  
 	
	
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here

if (hipMemcpy(deviceInput1,hostInput1,inputLength* sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  printf("data transfer error from host to device on deviceInput1\n");
  return 0;
 }

if (hipMemcpy(deviceInput2,hostInput2,inputLength* sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  printf("data transfer error from host to device on deviceInput1\n");
  return 0;
 }

/* Don't think we have to do this
if (hipMemcpy(deviceOutput,hostOutput,inputLength* sizeof(float),hipMemcpyHostToDevice) != hipSuccess){
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  printf("data transfer error from host to device on deviceInput1\n");
  return 0;
 }
*/
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here

dim3 mygrid(ceil(inputLength/256.0));
dim3 myblock(256);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here

  vecAdd<<<mygrid,myblock>>>(deviceInput1,deviceInput2,deviceOutput, inputLength);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here

  /*if (hipMemcpy(hostOutput,deviceOutput,inputLength*sizeof(float),hipMemcpyDeviceToHost) != hipSuccess){

   hipFree(deviceInput1);
   hipFree(deviceInput2);
   printf("data transfer error from host to device on deviceOutput\n");

   return 0;
 }*/

  hipMemcpy(hostOutput,deviceOutput,inputLength*sizeof(float),hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);	
  	

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
