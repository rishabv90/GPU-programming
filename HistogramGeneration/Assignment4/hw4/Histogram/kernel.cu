
#include <hip/hip_runtime.h>
// version 0
// global memory only interleaved version
// include comments describing your approach
__global__ void histogram_global_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

// insert your code here

}


// version 1
// shared memory privatized version
// include comments describing your approach
__global__ void histogram_shared_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

// insert your code here

}


// version 2
// your method of optimization using shared memory 
// include DETAILED comments describing your approach
__global__ void histogram_shared_accumulate_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

// insert your code here

}

// clipping function
// resets bins that have value larger than 127 to 127. 
// that is if bin[i]>127 then bin[i]=127

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {


// insert your code here

}
