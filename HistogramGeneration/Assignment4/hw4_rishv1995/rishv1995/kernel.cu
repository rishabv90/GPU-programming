#include "hip/hip_runtime.h"
#include <thrust/sort.h>



// version 0
// global memory only interleaved version
// include comments describing your approach
__global__ void histogram_global_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

// insert your code here
int i = blockIdx.x *blockDim.x + threadIdx.x; //Thread index
int stride = blockDim.x * gridDim.x; //Stride is total number of threads

while (i < num_elements) { //Loop
	int num_position = input[i]; // position on the array for bins
	if(num_position< num_bins && num_position >= 0) { //boundary check for number in bin allocation
		atomicAdd(&bins[num_position], 1); //increment bins	
	}
	i = i + stride; // increment loop counter
}

}


// version 1
// shared memory privatized version
// include comments describing your approach
__global__ void histogram_shared_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

// insert your code here

int i = blockIdx.x *blockDim.x + threadIdx.x; //Thread index
int stride = blockDim.x * gridDim.x; //Stride is total number of threads
__shared__ unsigned int histo_private[4096];//number of histogram privatized bins

if(threadIdx.x < 4096){ 
	histo_private[threadIdx.x]= 0; //intialize each bin to 0
} 
__syncthreads();// syncronize threads in a block

while (i < num_elements) { //Loop
	int num_position = input[i]; //position on the array for bins
	if(num_position < 4096 && num_position >=0){//boundary check for number in bin allocation
		atomicAdd(&histo_private[num_position], 1);
	}
	i = i + stride;	//incremetn loop counter	
}

__syncthreads();// syncronize threads in a block barrier sync


int j = 0; //counter for global histogram

while (j < num_bins) { //loop to atomicadd on bins since they are more than the size of the block....
	atomicAdd(&bins[threadIdx.x + j], histo_private[threadIdx.x + j]);
	j = j + blockDim.x; //increment loop counter
}

}



// version 2
// your method of optimization using shared memory 
// include DETAILED comments describing your approach
__global__ void histogram_shared_accumulate_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

// insert your code here


//unable to utilize thrust sorting and reduce by key feature as I am not familiar with the library.
//intented to sort privatised histogram before running the last loop or sorting bins array before developing histo_private. 
int i = blockIdx.x *blockDim.x + threadIdx.x; //Thread index
int stride = blockDim.x * gridDim.x; //Stride is total number of threads
__shared__ unsigned int histo_private[4096];//number of histogram privatized bins

if(threadIdx.x < 4096){ 
	histo_private[threadIdx.x]= 0; //intialize each bin to 0
} 
__syncthreads();// syncronize threads in a block barrier sync

while (i < num_elements) { //Loop
	int num_position = input[i]; //position on the array for bins
	if(num_position < 4096 && num_position >=0){//boundary check for number in bin allocation
		atomicAdd(&histo_private[num_position], 1);
	}
	i = i + stride;	//incremetn loop counter	
}

__syncthreads();// syncronize threads in a block barrier sync

//sorting histo_private array 
/*
thrust::sort(A, A+N)
*/

//thrust::sort(thrust::device, bins, num_elements + bins);

int j = 0; //counter for global histogram

while (j < num_bins) { //loop to atomicadd on bins since they are more than the size of the block....
	atomicAdd(&bins[threadIdx.x + j], histo_private[threadIdx.x + j]);
	j = j + blockDim.x; //increment loop counter
}


}

// clipping function
// resets bins that have value larger than 127 to 127. 
// that is if bin[i]>127 then bin[i]=127

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {


// insert your code here
int i = blockIdx.x *blockDim.x + threadIdx.x; //Thread index
if (bins[i] > 127){ //limiting to 127 bins
	bins[i] = 127;
}

}


